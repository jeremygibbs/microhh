#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2017 Chiel van Heerwaarden
 * Copyright (c) 2011-2017 Thijs Heus
 * Copyright (c) 2014-2017 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstdio>
#include "grid.h"
#include "fields.h"
#include "thermo_buoy.h"
#include "master.h"
#include "finite_difference.h"
#include "tools.h"

namespace
{   
	__global__ 
    void calc_buoyancy_tend_2nd_g(double* __restrict__ wt, double* __restrict__ b, 
                                  int istart, int jstart, int kstart,
                                  int iend,   int jend,   int kend,
                                  int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
        const int k = blockIdx.z + kstart; 

        using Finite_difference::O2::interp2;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            wt[ijk] += interp2(b[ijk-kk], b[ijk]);
        }
    }
    
    __global__ 
    void calc_buoyancy_tend_u_2nd_g(double* const __restrict__ ut, const double* const __restrict__ b,
                                    const double sinalpha,
                                    const int istart, const int jstart, const int kstart,
                                    const int iend,   const int jend,   const int kend,
                                    const int jj,     const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
        const int k = blockIdx.z + kstart;

        const int ii1 = 1;

        using Finite_difference::O2::interp2;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            ut[ijk] += sinalpha * interp2(b[ijk-ii1], b[ijk]);
        }
    }
    
    __global__ 
    void calc_buoyancy_tend_w_2nd_g(double* __restrict__ wt, const double* const __restrict__ b,
                                    const double cosalpha,
                                    const int istart, const int jstart, const int kstart,
                                    const int iend, const int jend, const int kend,
                                    const int jj, const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
        const int k = blockIdx.z + kstart;

        const int kk1 = 1*kk;

        using Finite_difference::O2::interp2;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            wt[ijk] += cosalpha * interp2(b[ijk-kk1], b[ijk]);
        }
    }
    
    __global__ 
    void calc_buoyancy_tend_b_2nd_g(double* const __restrict__ bt,
                                    const double* const __restrict__ u, const double* const __restrict__ w,
                                    const double utrans, const double n2, const double sinalpha, const double cosalpha,
                                    const int istart, const int jstart, const int kstart,
                                    const int iend, const int jend, const int kend,
                                    const int jj, const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
        const int k = blockIdx.z + kstart;

        const int ii1 = 1;
        const int kk1 = 1*kk;

        using Finite_difference::O2::interp2;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            bt[ijk] -= n2 * ( sinalpha * ( interp2(u[ijk], u[ijk+ii1]) + utrans )
                            + cosalpha * ( interp2(w[ijk], w[ijk+kk1]) ) );
        }
    }
    
    __global__ 
    void calc_buoyancy_tend_4th_g(double* __restrict__ wt, double* __restrict__ b, 
                                  int istart, int jstart, int kstart,
                                  int iend,   int jend,   int kend,
                                  int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
        const int k = blockIdx.z + kstart;

        const int kk1 = 1*kk;
        const int kk2 = 2*kk;

        using namespace Finite_difference::O4;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            wt[ijk] += ci0*b[ijk-kk2] + ci1*b[ijk-kk1] + ci2*b[ijk] + ci3*b[ijk+kk1];
        }
    }
    
    __global__ 
    void calc_buoyancy_tend_u_4th_g(double* const __restrict__ ut, const double* const __restrict__ b,
                                    const double sinalpha,
                                    const int istart, const int jstart, const int kstart,
                                    const int iend,   const int jend,   const int kend,
                                    const int jj,     const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
        const int k = blockIdx.z + kstart;

        const int ii1 = 1;
        const int ii2 = 2;

        using namespace Finite_difference::O4;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            ut[ijk] += sinalpha * (ci0*b[ijk-ii2] + ci1*b[ijk-ii1] + ci2*b[ijk] + ci3*b[ijk+ii1]);
        }
    }

    __global__ 
    void calc_buoyancy_tend_w_4th_g(double* __restrict__ wt, const double* const __restrict__ b,
                                    const double cosalpha,
                                    const int istart, const int jstart, const int kstart,
                                    const int iend, const int jend, const int kend,
                                    const int jj, const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
        const int k = blockIdx.z + kstart;

        const int kk1 = 1*kk;
        const int kk2 = 2*kk;

        using namespace Finite_difference::O4;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            wt[ijk] += cosalpha * (ci0*b[ijk-kk2] + ci1*b[ijk-kk1] + ci2*b[ijk] + ci3*b[ijk+kk1]);
        }
    }

    __global__ 
    void calc_buoyancy_tend_b_4th_g(double* const __restrict__ bt,
                                    const double* const __restrict__ u, const double* const __restrict__ w,
                                    const double utrans, const double n2, const double sinalpha, const double cosalpha,
                                    const int istart, const int jstart, const int kstart,
                                    const int iend, const int jend, const int kend,
                                    const int jj, const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
        const int k = blockIdx.z + kstart;

        const int ii1 = 1;
        const int ii2 = 2;

        const int kk1 = 1*kk;
        const int kk2 = 2*kk;

        using namespace Finite_difference::O4;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            bt[ijk] -= n2 * ( sinalpha * ( (ci0*u[ijk-ii1] + ci1*u[ijk] + ci2*u[ijk+ii1] + ci3*u[ijk+ii2]) + utrans )
                            + cosalpha * (  ci0*w[ijk-kk1] + ci1*w[ijk] + ci2*w[ijk+kk1] + ci3*w[ijk+kk2]) );
        }
    }
    
    __global__ 
    void calc_buoyancy_g(double* __restrict__ b,double* __restrict__ bin, 
                         int istart, int jstart,
                         int iend,   int jend,   int kcells,
                         int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
        const int k = blockIdx.z; 

        if (i < iend && j < jend && k < kcells)
        {
            const int ijk = i + j*jj + k*kk;
            b[ijk] = bin[ijk];
        }
    }
    
    __global__ 
    void calc_buoyancy_bot_g(double* __restrict__ b,     double* __restrict__ bbot,
                             double* __restrict__ bin,    double* __restrict__ bbotin, 
                             int kstart, int icells, int jcells, int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x; 
        const int j = blockIdx.y*blockDim.y + threadIdx.y; 

        if (i < icells && j < jcells)
        {
            const int ij  = i + j*jj;
            const int ijk = i + j*jj + kstart*kk;

            bbot[ij] = bbotin[ij];
            b[ijk]   = bin[ijk];
        }
    }
    
    __global__ 
    void calc_buoyancy_flux_bot_g(double* __restrict__ bfluxbot, double* __restrict__ bfluxbotin,
                                  int kstart, int icells, int jcells, int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x; 
        const int j = blockIdx.y*blockDim.y + threadIdx.y; 

        if (i < icells && j < jcells)
        {
            const int ij  = i + j*jj;
            bfluxbot[ij] = bfluxbotin[ij];
        }
    }

    __global__ 
    void calc_N2_g(double* __restrict__ N2,    double* __restrict__ b,
                   const double bg_n2, double* __restrict__ dzi, 
                   int istart, int jstart, int kstart,
                   int iend,   int jend,   int kend,
                   int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
        const int k = blockIdx.z + kstart; 

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            N2[ijk] = 0.5*(b[ijk+kk] - b[ijk-kk])*dzi[k] + bg_n2;
        }
    }

} // End namespace.

#ifdef USECUDA
void Thermo_buoy::exec()
{
    const int blocki = grid->ithread_block;
    const int blockj = grid->jthread_block;
    const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
    const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, grid->kmax-1);
    dim3 blockGPU(blocki, blockj, 1);

    const int offs = grid->memoffset;
    
    if (grid->swspatialorder== "2")
    {
        if (has_slope || has_N2)
        {
	        const double sinalpha = std::sin(this->alpha);
            const double cosalpha = std::cos(this->alpha);
		    
            calc_buoyancy_tend_u_2nd_g<<<gridGPU, blockGPU>>>(
                &fields->ut->data_g[offs], &fields->sp["b"]->data_g[offs],
                sinalpha,
                grid->istart,  grid->jstart, grid->kstart,
                grid->iend,    grid->jend,   grid->kend,
                grid->icellsp, grid->ijcellsp);
            cuda_check_error(); 
		    
            calc_buoyancy_tend_w_2nd_g<<<gridGPU, blockGPU>>>(
                &fields->wt->data_g[offs], &fields->sp["b"]->data_g[offs],
                cosalpha,
                grid->istart,  grid->jstart, grid->kstart+1,
                grid->iend,    grid->jend,   grid->kend,
                grid->icellsp, grid->ijcellsp);
            cuda_check_error(); 
		    
            calc_buoyancy_tend_b_2nd_g<<<gridGPU, blockGPU>>>(
                &fields->st["b"]->data_g[offs],
                &fields->u->data_g[offs], &fields->w->data_g[offs],
                grid->utrans, n2, sinalpha, cosalpha,
                grid->istart,  grid->jstart, grid->kstart,
                grid->iend,    grid->jend,   grid->kend,
                grid->icellsp, grid->ijcellsp);
            cuda_check_error();
        }
        else 
        {
	        calc_buoyancy_tend_2nd_g<<<gridGPU, blockGPU>>>(
            &fields->wt->data_g[offs], &fields->sp["b"]->data_g[offs], 
            grid->istart,  grid->jstart, grid->kstart+1,
            grid->iend,    grid->jend,   grid->kend,
            grid->icellsp, grid->ijcellsp);
            cuda_check_error();
        }
    }
    else if (grid->swspatialorder== "4")
    {
        const double sinalpha = std::sin(this->alpha);
        const double cosalpha = std::cos(this->alpha);
        
        if (has_slope || has_N2)
        {
            calc_buoyancy_tend_u_4th_g<<<gridGPU, blockGPU>>>(
                &fields->ut->data_g[offs], &fields->sp["b"]->data_g[offs],
                sinalpha,
                grid->istart,  grid->jstart, grid->kstart,
                grid->iend,    grid->jend,   grid->kend,
                grid->icellsp, grid->ijcellsp);
            cuda_check_error(); 
		    
            calc_buoyancy_tend_w_4th_g<<<gridGPU, blockGPU>>>(
                &fields->wt->data_g[offs], &fields->sp["b"]->data_g[offs],
                cosalpha,
                grid->istart,  grid->jstart, grid->kstart+1,
                grid->iend,    grid->jend,   grid->kend,
                grid->icellsp, grid->ijcellsp);
            cuda_check_error(); 
		    
            calc_buoyancy_tend_b_4th_g<<<gridGPU, blockGPU>>>(
                &fields->st["b"]->data_g[offs],
                &fields->u->data_g[offs], &fields->w->data_g[offs],
                grid->utrans, n2, sinalpha, cosalpha,
                grid->istart,  grid->jstart, grid->kstart,
                grid->iend,    grid->jend,   grid->kend,
                grid->icellsp, grid->ijcellsp);
            cuda_check_error();
        }
        else
        {
	        calc_buoyancy_tend_4th_g<<<gridGPU, blockGPU>>>(
            &fields->wt->data_g[offs], &fields->sp["b"]->data_g[offs], 
            grid->istart,  grid->jstart, grid->kstart+1,
            grid->iend,    grid->jend,   grid->kend,
            grid->icellsp, grid->ijcellsp);
            cuda_check_error();
        }
    }
}
#endif

#ifdef USECUDA
void Thermo_buoy::get_thermo_field(Field3d *fld, Field3d *tmp, std::string name, bool cyclic)
{
    const int blocki = grid->ithread_block;
    const int blockj = grid->jthread_block;
    const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
    const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);
    const double n2 = this->n2;
    
    dim3 gridGPU (gridi, gridj, grid->kcells);
    dim3 blockGPU(blocki, blockj, 1);

    dim3 gridGPU2 (gridi, gridj, grid->kmax);
    dim3 blockGPU2(blocki, blockj, 1);

    const int offs = grid->memoffset;

    if (name == "b")
    {
        calc_buoyancy_g<<<gridGPU, blockGPU>>>(
            &fld->data_g[offs], &fields->sp["b"]->data_g[offs], 
            grid->istart, grid->jstart, 
            grid->iend, grid->jend, grid->kcells,
            grid->icellsp, grid->ijcellsp);
        cuda_check_error();
    }
    else if (name == "N2")
    {
        calc_N2_g<<<gridGPU2, blockGPU2>>>(
            &fld->data_g[offs], &fields->sp["b"]->data_g[offs], n2, grid->dzi_g, 
            grid->istart,  grid->jstart, grid->kstart, 
            grid->iend,    grid->jend,   grid->kend,
            grid->icellsp, grid->ijcellsp);
        cuda_check_error();
    }
    else
    {
        master->print_error("get_thermo_field \"%s\" not supported\n",name.c_str());
        throw 1;
    }

    if (cyclic)
        grid->boundary_cyclic_g(&fld->data_g[offs]);
}
#endif

#ifdef USECUDA
void Thermo_buoy::get_buoyancy_fluxbot(Field3d *bfield)
{
    const int blocki = grid->ithread_block;
    const int blockj = grid->jthread_block;
    const int gridi  = grid->icells/blocki + (grid->icells%blocki > 0);
    const int gridj  = grid->jcells/blockj + (grid->jcells%blockj > 0);

    dim3 gridGPU (gridi, gridj, 1);
    dim3 blockGPU(blocki, blockj, 1);

    const int offs = grid->memoffset;

    calc_buoyancy_flux_bot_g<<<gridGPU, blockGPU>>>(
        &bfield->datafluxbot_g[offs], &fields->sp["b"]->datafluxbot_g[offs], 
        grid->kstart, grid->icells, grid->jcells,grid->icellsp, grid->ijcellsp);
    cuda_check_error();
}
#endif

#ifdef USECUDA
void Thermo_buoy::get_buoyancy_surf(Field3d *bfield)
{
    const int blocki = grid->ithread_block;
    const int blockj = grid->jthread_block;
    const int gridi  = grid->icells/blocki + (grid->icells%blocki > 0);
    const int gridj  = grid->jcells/blockj + (grid->jcells%blockj > 0);

    dim3 gridGPU (gridi, gridj, 1);
    dim3 blockGPU(blocki, blockj, 1);

    const int offs = grid->memoffset;

    calc_buoyancy_bot_g<<<gridGPU, blockGPU>>>(
        &bfield->data_g[offs], &bfield->databot_g[offs], 
        &fields->sp["b"]->data_g[offs], &fields->sp["b"]->databot_g[offs],
        grid->kstart, grid->icells, grid->jcells, 
        grid->icellsp, grid->ijcellsp);
    cuda_check_error();

    calc_buoyancy_flux_bot_g<<<gridGPU, blockGPU>>>(
        &bfield->datafluxbot_g[offs], &fields->sp["b"]->datafluxbot_g[offs], 
        grid->kstart, grid->icells, grid->jcells, grid->icellsp, grid->ijcellsp);
    cuda_check_error();
}
#endif