#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2017 Chiel van Heerwaarden
 * Copyright (c) 2011-2017 Thijs Heus
 * Copyright (c) 2014-2017 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <iostream>
#include "master.h"
#include "force.h"
#include "grid.h"
#include "fields.h"
#include "finite_difference.h"
#include "constants.h"
#include "tools.h"
#include "boundary.h"
#include "model.h"
#include "timeloop.h"

using namespace Finite_difference::O2;

namespace
{
    __global__ 
    void flux_step_1_g(double* const __restrict__ aSum, const double* const __restrict__ a,
                       const double* const __restrict__ dz,
                       const int jj, const int kk, 
                       const int istart, const int jstart, const int kstart,
                       const int iend,   const int jend,   const int kend)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            aSum [ijk] = a[ijk]*dz[k];
        }
    }

    __global__ 
    void flux_step_2_g(double* const __restrict__ ut,
                       const double fbody,
                       const int jj, const int kk, 
                       const int istart, const int jstart, const int kstart,
                       const int iend,   const int jend,   const int kend)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            ut[ijk] += fbody;
        }
    }

    __global__ 
    void coriolis_2nd_g(double* const __restrict__ ut, double* const __restrict__ vt,
                        double* const __restrict__ u,  double* const __restrict__ v, 
                        double* const __restrict__ ug, double* const __restrict__ vg, 
                        const double fc, const double ugrid, const double vgrid,
                        const int jj, const int kk, 
                        const int istart, const int jstart, const int kstart,
                        const int iend,   const int jend,   const int kend)
    {
        const int i  = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j  = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k  = blockIdx.z + kstart;
        const int ii = 1;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            ut[ijk] += fc * (0.25*(v[ijk-ii] + v[ijk] + v[ijk-ii+jj] + v[ijk+jj]) + vgrid - vg[k]);
            vt[ijk] -= fc * (0.25*(u[ijk-jj] + u[ijk] + u[ijk+ii-jj] + u[ijk+ii]) + ugrid - ug[k]);
        }
    }

    __global__ 
    void coriolis_4th_g(double* const __restrict__ ut, double* const __restrict__ vt,
                        double* const __restrict__ u,  double* const __restrict__ v, 
                        double* const __restrict__ ug, double* const __restrict__ vg, 
                        const double fc, const double ugrid, const double vgrid,
                        const int jj, const int kk, 
                        const int istart, const int jstart, const int kstart,
                        const int iend,   const int jend,   const int kend)
    {
        using namespace Finite_difference::O4;

        const int i   = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j   = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k   = blockIdx.z + kstart;
        const int ii  = 1;
        const int ii2 = 2;
        const int jj2 = 2*jj;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            ut[ijk] += fc * ( ( ci0*(ci0*v[ijk-ii2-jj ] + ci1*v[ijk-ii-jj ] + ci2*v[ijk-jj    ] + ci3*v[ijk+ii-jj  ])
                              + ci1*(ci0*v[ijk-ii2    ] + ci1*v[ijk-ii    ] + ci2*v[ijk       ] + ci3*v[ijk+ii     ])
                              + ci2*(ci0*v[ijk-ii2+jj ] + ci1*v[ijk-ii+jj ] + ci2*v[ijk+jj    ] + ci3*v[ijk+ii+jj  ])
                              + ci3*(ci0*v[ijk-ii2+jj2] + ci1*v[ijk-ii+jj2] + ci2*v[ijk+jj2   ] + ci3*v[ijk+ii+jj2 ]) )
                       + vgrid - vg[k] );

            vt[ijk] -= fc * ( ( ci0*(ci0*u[ijk-ii-jj2 ] + ci1*u[ijk-jj2   ] + ci2*u[ijk+ii-jj2] + ci3*u[ijk+ii2-jj2])
                              + ci1*(ci0*u[ijk-ii-jj  ] + ci1*u[ijk-jj    ] + ci2*u[ijk+ii-jj ] + ci3*u[ijk+ii2-jj ])
                              + ci2*(ci0*u[ijk-ii     ] + ci1*u[ijk       ] + ci2*u[ijk+ii    ] + ci3*u[ijk+ii2    ])
                              + ci3*(ci0*u[ijk-ii+jj  ] + ci1*u[ijk+jj    ] + ci2*u[ijk+ii+jj ] + ci3*u[ijk+ii2+jj ]) )
                       + ugrid - ug[k]);
        }
    }

    __global__ 
    void advec_wls_2nd_g(double* const __restrict__ st, double* const __restrict__ s,
                         const double* const __restrict__ wls, const double* const __restrict__ dzhi,
                         const int istart, const int jstart, const int kstart,
                         const int iend,   const int jend,   const int kend,
                         const int jj,     const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;

            if (wls[k] > 0.)
                st[ijk] -=  wls[k] * (s[k]-s[k-1])*dzhi[k];
            else
                st[ijk] -=  wls[k] * (s[k+1]-s[k])*dzhi[k+1];
        }
    }

    __global__ 
    void large_scale_source_g(double* const __restrict__ st, double* const __restrict__ sls,
                              const int istart, const int jstart, const int kstart,
                              const int iend,   const int jend,   const int kend,
                              const int jj,     const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            st[ijk] += sls[k];
        }
    }

    __global__ 
    void nudging_tendency_g(double* const __restrict__ st, double* const __restrict__ smn,
			    double* const __restrict__ snudge, double* const __restrict__ nudge_fac,
                            const int istart, const int jstart, const int kstart,
                            const int iend,   const int jend,   const int kend,
                            const int jj,     const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;

            st[ijk] += - nudge_fac[k] * (smn[k]-snudge[k]);

        }
    }

    __global__ 
    void update_time_dependent_prof_g(double* const __restrict__ prof, const double* const __restrict__ data,
                                      const double fac0, const double fac1, 
                                      const int index0,  const int index1, 
                                      const int kmax,    const int kgc)
    {
        const int k = blockIdx.x*blockDim.x + threadIdx.x;
        const int kk = kmax;

        if (k < kmax)
            prof[k+kgc] = fac0*data[index0*kk+k] + fac1*data[index1*kk+k];
    }
} // end namespace

void Force::prepare_device()
{
    const int nmemsize  = grid->kcells*sizeof(double);

    if (swlspres == "geo")
    {
        cuda_safe_call(hipMalloc(&ug_g, nmemsize));
        cuda_safe_call(hipMalloc(&vg_g, nmemsize));

        cuda_safe_call(hipMemcpy(ug_g, ug, nmemsize, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(vg_g, vg, nmemsize, hipMemcpyHostToDevice));
        if (swtimedep_geo == "1")
        {

            for (std::map<std::string, double *>::const_iterator it=timedepdata_geo.begin(); it!=timedepdata_geo.end(); ++it)
            {
                int nmemsize2 = grid->kmax*timedeptime_geo[it->first].size()*sizeof(double);
                cuda_safe_call(hipMalloc(&timedepdata_geo_g[it->first], nmemsize2));
                cuda_safe_call(hipMemcpy(timedepdata_geo_g[it->first], timedepdata_geo[it->first], nmemsize2, hipMemcpyHostToDevice));
            }
        }
    }

    if (swls == "1")
    {
        for (std::vector<std::string>::const_iterator it=lslist.begin(); it!=lslist.end(); ++it)
        {
            cuda_safe_call(hipMalloc(&lsprofs_g[*it], nmemsize));
            cuda_safe_call(hipMemcpy(lsprofs_g[*it], lsprofs[*it], nmemsize, hipMemcpyHostToDevice));
        }
        if (swtimedep_ls == "1")
        {
            for (std::map<std::string, double *>::const_iterator it=timedepdata_ls.begin(); it!=timedepdata_ls.end(); ++it)
            {
                int nmemsize2 = grid->kmax*timedeptime_ls[it->first].size()*sizeof(double);
                cuda_safe_call(hipMalloc(&timedepdata_ls_g[it->first], nmemsize2));
                cuda_safe_call(hipMemcpy(timedepdata_ls_g[it->first], timedepdata_ls[it->first], nmemsize2, hipMemcpyHostToDevice));
            }
        }
    }

    if (swnudge == "1")
    {
        for (std::vector<std::string>::const_iterator it=nudgelist.begin(); it!=nudgelist.end(); ++it)
        {
            cuda_safe_call(hipMalloc(&nudgeprofs_g[*it], nmemsize));
            cuda_safe_call(hipMemcpy(nudgeprofs_g[*it], nudgeprofs[*it], nmemsize, hipMemcpyHostToDevice));
        }
        cuda_safe_call(hipMalloc(&nudge_factor_g, nmemsize));
        cuda_safe_call(hipMemcpy(nudge_factor_g, nudge_factor, nmemsize, hipMemcpyHostToDevice));
        if (swtimedep_nudge == "1")
        {
            for (std::map<std::string, double *>::const_iterator it=timedepdata_nudge.begin(); it!=timedepdata_nudge.end(); ++it)
            {
                int nmemsize2 = grid->kmax*timedeptime_nudge[it->first].size()*sizeof(double);
                cuda_safe_call(hipMalloc(&timedepdata_nudge_g[it->first], nmemsize2));
                cuda_safe_call(hipMemcpy(timedepdata_nudge_g[it->first], timedepdata_nudge[it->first], nmemsize2, hipMemcpyHostToDevice));
            }
        }
    }

    if (swwls == "1")
    {
        cuda_safe_call(hipMalloc(&wls_g, nmemsize));
        cuda_safe_call(hipMemcpy(wls_g, wls, nmemsize, hipMemcpyHostToDevice));
        if (swtimedep_wls == "1")
        {
            int nmemsize2 = grid->kmax*timedeptime_wls.size()*sizeof(double);
            cuda_safe_call(hipMalloc(&timedepdata_wls_g, nmemsize2));
            cuda_safe_call(hipMemcpy(timedepdata_wls_g, timedepdata_wls, nmemsize2, hipMemcpyHostToDevice));
        }
    }

}

void Force::clear_device()
{
    if (swlspres == "geo")
    {
        cuda_safe_call(hipFree(ug_g));
        cuda_safe_call(hipFree(vg_g));
        if (swtimedep_geo == "1")
        {
            for (std::map<std::string, double *>::const_iterator it=timedepdata_geo.begin(); it!=timedepdata_geo.end(); ++it)
                cuda_safe_call(hipFree(timedepdata_geo_g[it->first]));
        }
    }

    if (swls == "1")
    {
        for(std::vector<std::string>::const_iterator it=lslist.begin(); it!=lslist.end(); ++it)
            cuda_safe_call(hipFree(lsprofs_g[*it]));
        if (swtimedep_ls == "1")
        {
            for (std::map<std::string, double *>::const_iterator it=timedepdata_ls.begin(); it!=timedepdata_ls.end(); ++it)
                cuda_safe_call(hipFree(timedepdata_ls_g[it->first]));
        }
    }

    if (swnudge == "1")
    {
        for(std::vector<std::string>::const_iterator it=nudgelist.begin(); it!=nudgelist.end(); ++it)
            cuda_safe_call(hipFree(nudgeprofs_g[*it]));
        cuda_safe_call(hipFree(nudge_factor_g));
        if (swtimedep_nudge == "1")
        {
            for (std::map<std::string, double *>::const_iterator it=timedepdata_nudge.begin(); it!=timedepdata_nudge.end(); ++it)
                cuda_safe_call(hipFree(timedepdata_nudge_g[it->first]));
        }
    }

    if (swwls == "1")
    {
        cuda_safe_call(hipFree(wls_g));
        if (swtimedep_wls == "1")
        {
            cuda_safe_call(hipFree(timedepdata_wls_g));
        }
    }
}

#ifdef USECUDA
void Force::exec(double dt)
{
    const int blocki = grid->ithread_block;
    const int blockj = grid->jthread_block;
    const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
    const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, grid->kcells);
    dim3 blockGPU(blocki, blockj, 1);

    const int offs = grid->memoffset;

    if (swlspres == "uflux")
    {
        flux_step_1_g<<<gridGPU, blockGPU>>>(
            &fields->atmp["tmp1"]->data_g[offs], &fields->u->data_g[offs],
            grid->dz_g,
            grid->icellsp, grid->ijcellsp,
            grid->istart,  grid->jstart, grid->kstart,
            grid->iend,    grid->jend,   grid->kend);
        cuda_check_error();

    double uavg  = grid->get_sum_g(&fields->atmp["tmp1"]->data_g[offs], fields->atmp["tmp2"]->data_g); 

        flux_step_1_g<<<gridGPU, blockGPU>>>(
            &fields->atmp["tmp1"]->data_g[offs], &fields->ut->data_g[offs],
            grid->dz_g,
            grid->icellsp, grid->ijcellsp,
            grid->istart,  grid->jstart, grid->kstart,
            grid->iend,    grid->jend,   grid->kend);
        cuda_check_error();

    double utavg = grid->get_sum_g(&fields->atmp["tmp1"]->data_g[offs], fields->atmp["tmp2"]->data_g); 

        uavg  = uavg  / (grid->itot*grid->jtot*grid->zsize);
        utavg = utavg / (grid->itot*grid->jtot*grid->zsize);

        const double fbody = (uflux - uavg - grid->utrans) / dt - utavg;

        flux_step_2_g<<<gridGPU, blockGPU>>>(
            &fields->ut->data_g[offs],
            fbody,
            grid->icellsp, grid->ijcellsp,
            grid->istart,  grid->jstart, grid->kstart,
            grid->iend,    grid->jend,   grid->kend);
        cuda_check_error();
    }
    else if (swlspres == "dpdxls")
    {
        flux_step_2_g<<<gridGPU, blockGPU>>>(
            &fields->ut->data_g[offs],
            dpdxls,
            grid->icellsp, grid->ijcellsp,
            grid->istart,  grid->jstart, grid->kstart,
            grid->iend,    grid->jend,   grid->kend);
        cuda_check_error();
    }
    else if (swlspres == "geo")
    {
        if (grid->swspatialorder == "2")
        {
            coriolis_2nd_g<<<gridGPU, blockGPU>>>(
                &fields->ut->data_g[offs], &fields->vt->data_g[offs],
                &fields->u->data_g[offs],  &fields->v->data_g[offs],
                ug_g, vg_g, fc, grid->utrans, grid->vtrans, 
                grid->icellsp, grid->ijcellsp,
                grid->istart,  grid->jstart, grid->kstart,
                grid->iend,    grid->jend,   grid->kend);
            cuda_check_error();
        }
        else if (grid->swspatialorder == "4")
        {
            coriolis_4th_g<<<gridGPU, blockGPU>>>(
                &fields->ut->data_g[offs], &fields->vt->data_g[offs],
                &fields->u->data_g[offs],  &fields->v->data_g[offs],
                ug_g, vg_g, fc, grid->utrans, grid->vtrans, 
                grid->icellsp, grid->ijcellsp,
                grid->istart,  grid->jstart, grid->kstart,
                grid->iend,    grid->jend,   grid->kend);
            cuda_check_error();
        }
    }

    if (swls == "1")
    {
        for (std::vector<std::string>::const_iterator it=lslist.begin(); it!=lslist.end(); ++it)
        {
            large_scale_source_g<<<gridGPU, blockGPU>>>(
                &fields->st[*it]->data_g[offs], lsprofs_g[*it],
                grid->istart,  grid->jstart, grid->kstart,
                grid->iend,    grid->jend,   grid->kend,
                grid->icellsp, grid->ijcellsp);
            cuda_check_error();
        }
    }

    if (swnudge == "1")
    {
        for(std::vector<std::string>::const_iterator it=nudgelist.begin(); it!=nudgelist.end(); ++it)
        {
            nudging_tendency_g<<<gridGPU, blockGPU>>>(
                &fields->at[*it]->data_g[offs],  fields->ap[*it]->datamean_g, 
                nudgeprofs_g[*it], nudge_factor_g,
                grid->istart,  grid->jstart, grid->kstart,
                grid->iend,    grid->jend,   grid->kend,
                grid->icellsp, grid->ijcellsp);
            cuda_check_error();
        }
    } 
    
    if (swwls == "1")
    {
        for (FieldMap::iterator it = fields->st.begin(); it!=fields->st.end(); it++)
        {
            advec_wls_2nd_g<<<gridGPU, blockGPU>>>(
                &it->second->data_g[offs], fields->sp[it->first]->datamean_g, wls_g, grid->dzhi_g,
                grid->istart,  grid->jstart, grid->kstart,
                grid->iend,    grid->jend,   grid->kend,
                grid->icellsp, grid->ijcellsp);
            cuda_check_error();
        }
    }
}
#endif

#ifdef USECUDA
void Force::update_time_dependent_profs(std::map<std::string, double*>& profiles, std::map<std::string, double*> time_profiles,
                                        std::map<std::string, std::vector<double>> times, std::string suffix)
{
    const int blockk = 128;
    const int gridk  = grid->kmax/blockk + (grid->kmax%blockk > 0);

    // Loop over all profiles which might be time dependent
    for (auto& it : profiles)
    {
        std::string name = it.first + suffix;

        // Check if they have time dependent data
        if (time_profiles.find(name) != time_profiles.end())
        {
            // Get/calculate the interpolation indexes/factors
            int index0, index1;
            double fac0, fac1;

            model->timeloop->get_interpolation_factors(index0, index1, fac0, fac1, times[name]);

            // Calculate the new vertical profile
            update_time_dependent_prof_g<<<gridk, blockk>>>(
                it.second, time_profiles[name], fac0, fac1, index0, index1, grid->kmax, grid->kgc);
            cuda_check_error();
        }
    }

}
#endif

#ifdef USECUDA
void Force::update_time_dependent_prof(double* const prof, const double* const data, std::vector<double> times)
{
    const int blockk = 128;
    const int gridk  = grid->kmax/blockk + (grid->kmax%blockk > 0);

    int index0, index1;
    double fac0, fac1;

    model->timeloop->get_interpolation_factors(index0, index1, fac0, fac1, times);

    // Calculate the new vertical profile
    update_time_dependent_prof_g<<<gridk, blockk>>>(
        prof, data, fac0, fac1, index0, index1, grid->kmax, grid->kgc);
    cuda_check_error();

}
#endif
